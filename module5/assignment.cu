//
//  assignment4.cu
//  assignment4
//
//  Created by Wilson on 2/18/22.
//


#include <iostream>
#include <chrono>
#include <vector>
#include "Utilities.h"
#include "ADD.h"
#include "SUB.h"
#include "MUL.h"
#include "MOD.h"

using namespace std;
using namespace std::chrono;



void run_Funs(UInt32 *gpu_arr1, UInt32 *gpu_arr2, 
         UInt32 numBlocks, UInt32 blockSize)
{
	RESULT addR, subR, mulR, modR; 
	const UInt32 ARRAY_SIZE = numBlocks * blockSize;
	
	//Do the four mathematical calculation and output
	//the result
	Topadd(gpu_arr1, gpu_arr2, numBlocks, blockSize, &addR);
	Topsub(gpu_arr1, gpu_arr2, numBlocks, blockSize, &subR);
	Topmul(gpu_arr1, gpu_arr2, numBlocks, blockSize, &mulR);
	Topmod(gpu_arr1, gpu_arr2, numBlocks, blockSize, &modR); 
	hipDeviceSynchronize();
	output(gpu_arr1, gpu_arr2, &addR, &subR, &mulR, &modR, ARRAY_SIZE);
}

void main_Pegeable(UInt32 totalThreads, UInt32 numBlocks, 
				   UInt32 blockSize)
{
	const UInt32 ARRAY_SIZE = totalThreads;
	UInt32 ARRAY_SIZE_IN_BYTES  = (sizeof(UInt32) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	UInt32 *cpu_arr1, *cpu_arr2, *gpu_arr1, *gpu_arr2;

	cpu_arr1 = (UInt32 *)malloc(ARRAY_SIZE_IN_BYTES);
	cpu_arr2 = (UInt32 *)malloc(ARRAY_SIZE_IN_BYTES);	
	hipMalloc((void **)&gpu_arr1,      ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2,      ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
					  
	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	
	
	//free GPU and CPU memory
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	free(cpu_arr1);
	free(cpu_arr2);
}

void main_Pinned(UInt32 totalThreads, UInt32 numBlocks, 
				 UInt32 blockSize)
{
	const UInt32 ARRAY_SIZE = totalThreads;
	UInt32 ARRAY_SIZE_IN_BYTES  = (sizeof(UInt32) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	UInt32 *cpu_arr1, *cpu_arr2, *gpu_arr1, *gpu_arr2;

	hipHostMalloc((UInt32 **)&cpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipHostMalloc((UInt32 **)&cpu_arr2, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2, ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);

	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	

	//free GPU and CPU memory
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipHostFree(cpu_arr1);
	hipHostFree(cpu_arr2);
}

int main(int argc, char** argv)
{
	// read command line arguments
	UInt32 totalThreads = 64;
	UInt32 blockSize    = 32;
	UInt32 numBlocks    = 8;
	
	if (argc >= 2) {
        
        sscanf(argv[1], "%d", &totalThreads);
	}
	if (argc >= 3) {
        sscanf(argv[2], "%d", &blockSize);
	}

	numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) 
	{
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		cout<<"Warning: Total thread count is not evenly divisible by the block size\n";
		cout<<"The total number of threads will be rounded up to "<< totalThreads<<endl;
	}

	//launch the main_Pegleble() or main_main_Pinned()
	// and measure the execution time
	float delta1 = 0, delta2 = 0;
	hipEvent_t start = get_time();	
	main_Pegeable(totalThreads, numBlocks, blockSize); 
	hipEvent_t stop = get_time();	
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&delta1, start, stop);

	start = get_time();	
	main_Pinned(totalThreads, numBlocks, blockSize); 
	stop = get_time();	
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&delta2, start, stop);
	outputTime(delta1,delta2);


	return EXIT_SUCCESS;
}