//
//  assignment3.cu
//  assignment3
//
//  Created by Wilson on 2/10/22.
//


#include <iostream>
#include <chrono>
#include <vector>
#include "Utilities.h"
#include "ADD.h"

using namespace std;
using namespace std::chrono;



void run_Funs(unsigned int *gpu_arr1, unsigned int *gpu_arr2, 
         unsigned int numBlocks, unsigned int blockSize)
{
	RESULT addR; const unsigned int ARRAY_SIZE = numBlocks * blockSize;
	
	Topadd(gpu_arr1, gpu_arr1, numBlocks, blockSize, &addR);
	output(&addR, ARRAY_SIZE);

}

void main_Pegeable(unsigned int totalThreads, unsigned int  blockSize, unsigned int numBlocks)
{
	const unsigned int ARRAY_SIZE = totalThreads;
	unsigned int ARRAY_SIZE_IN_BYTES  = (sizeof(unsigned int) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	unsigned int *cpu_arr1, *cpu_arr2;

	cpu_arr1 = (unsigned int *)malloc(ARRAY_SIZE_IN_BYTES);
	cpu_arr2 = (unsigned int *)malloc(ARRAY_SIZE_IN_BYTES);
	
	/* Declare pointers for GPU based params */
	unsigned int *gpu_arr1;
	unsigned int *gpu_arr2;
	
	hipMalloc((void **)&gpu_arr1,      ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2,      ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
					  
	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	

	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	free(cpu_arr1);
	free(cpu_arr2);
}

void main_Pinned(unsigned int totalThreads, unsigned int  blockSize, unsigned int numBlocks)
{
	const unsigned int ARRAY_SIZE = totalThreads;
	unsigned int ARRAY_SIZE_IN_BYTES  = (sizeof(unsigned int) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	unsigned int *cpu_arr1, *cpu_arr2;

	hipHostAlloc((unsigned int *)&cpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipHostAlloc((unsigned int *)&cpu_arr2, ARRAY_SIZE_IN_BYTES);

	/* Declare pointers for GPU based params */
	unsigned int *gpu_arr1;
	unsigned int *gpu_arr2;
	
	hipMalloc((void **)&gpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2, ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
					  
	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	

	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipHostFree(cpu_arr1);
	hipHostFree(cpu_arr2);
}

int main(int argc, char** argv)
{
	// read command line arguments
	unsigned int totalThreads = 64;
	unsigned int blockSize    = 32;
	unsigned int numBlocks    = 8;
	
	if (argc >= 2) {
        
        sscanf(argv[1], "%d", &totalThreads);
	}
	if (argc >= 3) {
        sscanf(argv[2], "%d", &blockSize);
	}

	numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) 
	{
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		cout<<"Warning: Total thread count is not evenly divisible by the block size\n";
		cout<<"The total number of threads will be rounded up to "<< totalThreads<<endl;
	}
	
	main_Pegeable(totalThreads, blockSize, numBlocks);

	
	return EXIT_SUCCESS;
}