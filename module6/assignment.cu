//
//  assignment5.cu
//  assignment5
//
//  Created by Wilson on 2/25/22.
//


#include <iostream>
#include <chrono>
#include <vector>
#include "Utilities.h"
#include "ADD.h"
#include "SUB.h"
#include "MUL.h"
#include "MOD.h"

using namespace std;
using namespace std::chrono;



void run_Funs(UInt32 *gpu_arr1, UInt32 *gpu_arr2, 
         UInt32 numBlocks, UInt32 blockSize)
{
	RESULT addR, subR, mulR, modR; 
	const UInt32 ARRAY_SIZE = numBlocks * blockSize;
	
	//Do the four mathematical calculation and output
	//the result
	Topadd(gpu_arr1, gpu_arr2, numBlocks, blockSize, &addR);
	Topsub(gpu_arr1, gpu_arr2, numBlocks, blockSize, &subR);
	Topmul(gpu_arr1, gpu_arr2, numBlocks, blockSize, &mulR);
	Topmod(gpu_arr1, gpu_arr2, numBlocks, blockSize, &modR); 
	hipDeviceSynchronize();
	output(gpu_arr1, gpu_arr2, &addR, &subR, &mulR, &modR, ARRAY_SIZE);
}

void main_Pegeable(UInt32 totalThreads, UInt32 numBlocks, 
				   UInt32 blockSize)
{
	const UInt32 ARRAY_SIZE = totalThreads;
	UInt32 ARRAY_SIZE_IN_BYTES  = (sizeof(UInt32) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	UInt32 *cpu_arr1, *cpu_arr2, *gpu_arr1, *gpu_arr2;

	cpu_arr1 = (UInt32 *)malloc(ARRAY_SIZE_IN_BYTES);
	cpu_arr2 = (UInt32 *)malloc(ARRAY_SIZE_IN_BYTES);	
	hipMalloc((void **)&gpu_arr1,      ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2,      ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
					  
	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	
	
	//free GPU and CPU memory
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	free(cpu_arr1);
	free(cpu_arr2);
}

void main_Pinned(UInt32 totalThreads, UInt32 numBlocks, 
				 UInt32 blockSize)
{
	const UInt32 ARRAY_SIZE = totalThreads;
	UInt32 ARRAY_SIZE_IN_BYTES  = (sizeof(UInt32) * (ARRAY_SIZE));
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	UInt32 *cpu_arr1, *cpu_arr2, *gpu_arr1, *gpu_arr2;

	hipHostMalloc((UInt32 **)&cpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipHostMalloc((UInt32 **)&cpu_arr2, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2, ARRAY_SIZE_IN_BYTES);

	init(cpu_arr1, cpu_arr2, ARRAY_SIZE);	

	hipMemcpy(gpu_arr1, cpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr2, cpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);

	run_Funs(gpu_arr1, gpu_arr2, numBlocks, blockSize);	

	//free GPU and CPU memory
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);								  
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipHostFree(cpu_arr1);
	hipHostFree(cpu_arr2);
}

int main()
{
	
	UInt32 totalThreads = 12;
	UInt32 blockSize    = 12;
	UInt32 numBlocks    = 1;

	float delta1=0,delta2=0;
	UInt32 size[2];

	size[0] = totalThreads;
	hipEvent_t start = get_time();
	main_Pegeable(totalThreads, numBlocks, blockSize); 
	hipEvent_t stop = get_time();
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&delta1, start, stop);


	totalThreads = 14;
	blockSize    = 14;
	numBlocks    = 1;

	start = get_time();
	main_Pegeable(totalThreads, numBlocks, blockSize); 
	stop = get_time();
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&delta2, start, stop);

	size[1] = totalThreads;
	outputTime(delta1, delta2, size);

	return EXIT_SUCCESS;
}