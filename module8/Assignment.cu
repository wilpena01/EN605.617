#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>
#include <hipfft/hipfft.h>
#include "Utilities.h"

using namespace std;
using namespace std::chrono;

#define H  = 3;
#define W  = H;
#define HW = H*W;
typedef float2 Complex;

void mulMatAnalysis(float *A, float *B, float *C)
{
    
    float *h_A = (float*)malloc(HW*sizeof(float));
    float *h_B = (float*)malloc(HW*sizeof(float));
    float *h_C = (float*)malloc(HW*sizeof(float));

    equalMat(h_A,A,HW); equalMat(h_B,B,HW);
    
    auto start = high_resolution_clock::now();
    mulMat(h_A,h_B,H,W,h_C);
    auto stop = high_resolution_clock::now();
    auto duration1 = duration_cast<microseconds>(stop - start);

    float* g_A; float* g_B; float* g_C;

    /*ALLOCATE ON THE DEVICE*/
    cublasAlloc(HW,sizeof(float),(void**)&g_A);
    cublasAlloc(HW,sizeof(float),(void**)&g_B);
    cublasAlloc(HW,sizeof(float),(void**)&g_C);

    /*SET MATRIX*/
    hipblasSetMatrix(H,W,sizeof(float),A,H,g_A,H);
    hipblasSetMatrix(H,W,sizeof(float),B,H,g_B,H);
  
    /*KERNEL*/
    start = high_resolution_clock::now();
    hipblasSgemm('n','n',H,W,W,1,g_A,H,g_B,H,0,g_C,H);
    stop = high_resolution_clock::now();
    auto duration2 = duration_cast<microseconds>(stop - start);
    cublasGetError();
    hipblasGetMatrix(H,W,sizeof(float),g_C,H,C,H);

    /* PERFORMANCE OUTPUT*/

    printf("\nMatriz A:\n");
    printMat(A,W,H);
    printf("\nMatriz B:\n");
    printMat(B,W,H);
    printf("\nMatriz C:\n");
    printMat(C,W,H);

    free( h_A );  cublasFree(g_A);
    free( h_B );  cublasFree(g_B);
    free( h_C );  cublasFree(g_C);

    string str[] = {"cuBlas"};
    outputTime(duration1, duration2, str);
}

__global__ 
void ComplexMUL(Complex *mat1, Complex *mat2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    mat1[i].x = mat1[i].x * mat2[i].x - mat1[i].y*mat2[i].y;
    mat1[i].y = mat1[i].x * mat2[i].y + mat1[i].y*mat2[i].x;
}

void runcuFFT()
{
    Complex *fg = new Complex[HW];
    for (int i = 0; i < HW; i++){
        fg[i].x = 1;
        fg[i].y = 0;
    }
    Complex *fig = new Complex[HW];
    for (int i = 0; i < HW; i++){
        fig[i].x = 1; // 
        fig[i].y = 0;
    }
    for (int i = 0; i < H * W; i = i + H)
    {
        for (int j=0; j < W; j++){
            cout << fg[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;
    for (int i = 0; i < H * W; i = i + H)
    {
        for (int j=0; j < W; j++){
            cout << fig[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;

    int mem_size = sizeof(Complex)* HW;


    hipfftComplex *d_signal;
    hipMalloc((void **) &d_signal, mem_size); 
    hipMemcpy(d_signal, fg, mem_size, hipMemcpyHostToDevice);

    hipfftComplex *d_filter_kernel;
    hipMalloc((void **)&d_filter_kernel, mem_size);
    hipMemcpy(d_filter_kernel, fig, mem_size, hipMemcpyHostToDevice);

    // cout << d_signal[1].x << endl;
    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan2d(&plan, H, H, HIPFFT_C2C);

    // Transform signal and filter
    printf("Transforming signal hipfftExecR2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD);

    printf("Launching Complex multiplication<<< >>>\n");
    ComplexMUL <<< H, H >> >(d_signal, d_filter_kernel);

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    Complex *result = new Complex[HW];
    hipMemcpy(result, d_signal, sizeof(Complex)*HW, hipMemcpyDeviceToHost);

    for (int i = 0; i < HW; i = i + H)
    {
        for (int j=0; j < W; j++){
            cout << result[i+j].x << " ";
        }
        cout << endl;
    }

    delete result, fg, fig;
    hipfftDestroy(plan);
    //hipfftDestroy(plan2);
    hipFree(d_signal);
    hipFree(d_filter_kernel);

}

 int  main () 
 {
    cublasInit();

    float *A = (float*)malloc(HW*sizeof(float));
    float *B = (float*)malloc(HW*sizeof(float));
    float *C = (float*)malloc(HW*sizeof(float));

    initMat(A,H,W); 
    initMat(B,H,W); 

    mulMatAnalysis(A,B,C);
    runcuFFT();

    free( A ); 
    free( B ); 
    free( C );

    /* Shutdown */
    cublasShutdown();

		return 0;


  }
