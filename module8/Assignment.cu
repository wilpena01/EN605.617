
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <iostream>

using namespace std;

#define index(i,j,ld) (((j)*(ld))+(i))

void printMat(float*P,int uWP,int uHP){
  //printf("\n %f",P[1]);
  int i,j;
  for(i=0;i<uHP;i++){

      printf("\n");

      for(j=0;j<uWP;j++)
          printf("%f ",P[index(i,j,uHP)]);
  }
}

void mulMat(float *mat1, float* mat2,int H, int W, float *rslt ) 
{
 
    cout << "Multiplication of given two matrices is:\n" << endl;
 
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            rslt[index(i,j,H)] = 0;
 
            for (int k = 0; k < W; k++) {
                rslt[index(i,j,H)] += mat1[index(i,k,H)] * mat2[index(k,j,H)];
            }
 
            cout << rslt[index(i,j,H)] << "\t";
        }
 
        cout << endl;
    }
}


 int  main (int argc, char** argv) {
    hipblasStatus_t status;
    int i,j;
    cublasInit();

    int H = 9, W=9;

    float *A = (float*)malloc(H*W*sizeof(float));
    float *B = (float*)malloc(H*W*sizeof(float));
    float *C = (float*)malloc(H*W*sizeof(float));
    if (A == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (B == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (C == 0) {
      fprintf (stderr, "!!!! host memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    float *h_A = (float*)malloc(H*W*sizeof(float));
    float *h_B = (float*)malloc(H*W*sizeof(float));
    float *h_C = (float*)malloc(H*W*sizeof(float));


    for (i=0;i<H;i++)
      for (j=0;j<W;j++)
      {
        A[index(i,j,H)] = rand()%20; 
        h_A[index(i,j,H)] = A[index(i,j,H)]; 
      }
    for (i=0;i<H;i++)
      for (j=0;j<W;j++)
      {
        B[index(i,j,H)] = rand()%20; 
        h_B[index(i,j,H)] = B[index(i,j,H)]; 
      }

      mulMat(h_A,h_B,H,W,h_C);
 
    float* g_A; float* g_B; float* g_C;

    /*ALLOCATE ON THE DEVICE*/
    status=cublasAlloc(H*W,sizeof(float),(void**)&g_A);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=cublasAlloc(H*W,sizeof(float),(void**)&g_B);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=cublasAlloc(H*W,sizeof(float),(void**)&g_C);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    /*SET MATRIX*/
    status=hipblasSetMatrix(H,W,sizeof(float),A,H,g_A,H);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=hipblasSetMatrix(H,W,sizeof(float),B,H,g_B,H);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    /*KERNEL*/
    cublasSgemm('n','n',H,W,W,1,g_A,H,g_B,H,0,g_C,H);

    status = cublasGetError();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! kernel execution error.\n");
      return EXIT_FAILURE;
    }
    hipblasGetMatrix(H,W,sizeof(float),g_C,H,C,H);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device read error (A)\n");
      return EXIT_FAILURE;
    }


    /* PERFORMANCE OUTPUT*/

    printf("\nMatriz A:\n");
    printMat(A,W,H);
    printf("\nMatriz B:\n");
    printMat(B,W,H);
    printf("\nMatriz C:\n");
    printMat(C,W,H);

    free( A );  free( B );  free ( C );
    status = cublasFree(g_A);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (A)\n");
      return EXIT_FAILURE;
    }
    status = cublasFree(g_B);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (B)\n");
      return EXIT_FAILURE;
    }
    status = cublasFree(g_C);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (C)\n");
      return EXIT_FAILURE;
    }

    /* Shutdown */
    status = cublasShutdown();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! shutdown error (A)\n");
      return EXIT_FAILURE;
    }

    if (argc > 1) {
      if (!strcmp(argv[1], "-noprompt") ||!strcmp(argv[1], "-qatest") ){
        return EXIT_SUCCESS;
      }
    } 
    else{
      printf("\nPress ENTER to exit...\n");
      getchar();
    }

		return EXIT_SUCCESS;


  }
