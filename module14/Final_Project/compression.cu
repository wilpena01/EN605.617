// C Code for
// Image Compression
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "Utilities.h"
#include <string>
#include <fstream>


#include "ImagesCPU.h"
#include "ImagesNPP.h"
#include "ImageIO.h"
#include "Exceptions.h"
#include <npp.h>


#include "Compress_Helper.h"
#include "Compress_Helper_cu.h"


using namespace std;

// Driver code
void compressionDriver()
{
   int width, height;
   int** image;
   int hist[256];
   int nodes, maxcodelen, totalnodes;
   float p = 1.0; 
   pixfreq<25> *pix_freq;
   huffcode* huffcodes;

   readBMPFILE(width, height, image);
   ocurrence(hist, image, width, height);
   nonZero_ocurrence(hist, nodes);
   minProp(p, hist, width, height);
   maxcodelen = MaxLength(p) - 3;

   totalnodes = 2 * nodes - 1;
   pix_freq = (pixfreq<25>*)malloc(sizeof(pixfreq<25>) * totalnodes);
   huffcodes = (struct huffcode*)malloc(sizeof(struct huffcode) * nodes);

   InitStruct(pix_freq, huffcodes, hist, height, width);
   sortHist(huffcodes, nodes);
   BuildTree(pix_freq, huffcodes, nodes);
   AssignCode(pix_freq, nodes, totalnodes);
   PrintHuffmanCode(pix_freq, nodes);
   calBitLength(pix_freq, nodes);
   delete[] image; image = NULL;
}

void compressionDriver_CL()
{
   const int HistSize = 256;
   const int HistSize_Byte = sizeof(int) * HistSize;
   int width, height;
   int MaxSize;
   int* image;
   int* hist;
   int nodes = 0;
   int maxcodelen, totalnodes;
   float p = 1.0; 
   pixfreq<25> *pix_freq;
   huffcode* huffcodes;

   const int hist_num_blocks     = 1;
   const int hist_num_threads    = HistSize;

   const int image_num_blocks    = 512;
   const int image_num_threads   = 512;

   int* g_image;
   int* g_width, *g_height, *g_nodes, *g_totalnodes;
   int* g_MaxSize;
   int* g_hist;
   float* g_p;
   pixfreq<25>* g_pix_freq;
   huffcode* g_huffcodes;

   int *gpu_Result, *gpu_Block, *gpu_Thread;
   int *cpu_Result;
   int *cpu_Block;
   int *cpu_Thread;
   
   readBMPFILE_cu(width, height, image);
   MaxSize = width * height;
/*
   for(int i=0; i<height; i++)
   {
      for(int j=0; j<width; j++)
      {
         if(image[i][j]>256)
            cout<<"image ="<<image[i][j]<<"   ";
      }
   }

*/

   int IMAGE_SIZE_IN_BYTES = sizeof(int) * MaxSize;

   cpu_Result = (int *)malloc(HistSize_Byte);
	cpu_Block = (int *)malloc(HistSize_Byte);
   cpu_Thread = (int *)malloc(HistSize_Byte);
   int* image2 = (int *)malloc(IMAGE_SIZE_IN_BYTES);
   hist = (int *)malloc(HistSize_Byte);

   hipMalloc((void **)&g_image,       IMAGE_SIZE_IN_BYTES);
   hipMalloc((void **)&g_width,       sizeof(int));
   hipMalloc((void **)&g_height,      sizeof(int));
   hipMalloc((void **)&g_hist,        HistSize_Byte);
   hipMalloc((void **)&g_nodes,       sizeof(int));
   hipMalloc((void **)&g_p,           sizeof(float));
   hipMalloc((void **)&g_totalnodes,  sizeof(int));
   hipMalloc((void **)&g_MaxSize,     sizeof(int));
   hipMalloc((void **)&gpu_Result,    HistSize_Byte);
   hipMalloc((void **)&gpu_Block,     HistSize_Byte);
   hipMalloc((void **)&gpu_Thread,    HistSize_Byte);

   hipMemcpy(g_image,      image,       IMAGE_SIZE_IN_BYTES,   hipMemcpyHostToDevice);
   hipMemcpy(g_width,      &width,      sizeof(int),           hipMemcpyHostToDevice);
   hipMemcpy(g_height,     &height,     sizeof(int),           hipMemcpyHostToDevice);
   hipMemcpy(g_hist,       hist,        HistSize_Byte,         hipMemcpyHostToDevice);
   hipMemcpy(g_nodes,      &nodes,      sizeof(int),           hipMemcpyHostToDevice);
   hipMemcpy(g_p,          &p,          sizeof(int),           hipMemcpyHostToDevice);
   hipMemcpy(g_totalnodes, &totalnodes, sizeof(int),           hipMemcpyHostToDevice);
   hipMemcpy(g_MaxSize,    &MaxSize,    sizeof(int),           hipMemcpyHostToDevice);
   //hipMemcpy(gpu_Result,   cpu_Result,  HistSize_Byte,           hipMemcpyHostToDevice);
   //hipMemcpy(gpu_Block,    cpu_Block,   HistSize_Byte,           hipMemcpyHostToDevice);
  // hipMemcpy(gpu_Thread,   cpu_Thread,  HistSize_Byte,           hipMemcpyHostToDevice);


  // cout<<"heiht = "<<height<<"\twidth = "<<width<<endl;
   initHist_cu<<<hist_num_blocks, hist_num_threads>>>(g_hist, gpu_Result, gpu_Block, gpu_Thread);
               
               hipMemcpy(cpu_Result, gpu_Result, HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Block,  gpu_Block,  HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Thread, gpu_Thread, HistSize_Byte, hipMemcpyDeviceToHost);
              // outputResult(cpu_Result, cpu_Block, cpu_Thread, 256);



   hipMemcpy(image2,        g_image,       IMAGE_SIZE_IN_BYTES,  hipMemcpyDeviceToHost);
   
   for(int i=0; i<width*height; i++)
   {
         if(image2[i]==241)
            cout<<"image = "<<image2[i]<<"\n";
      
   }
   //cout<<"heiht = "<<height<<"\twidth = "<<width<<endl;

   //hipMemcpy(hist,        g_hist,       HistSize_Byte,  hipMemcpyDeviceToHost);

   //for(int i=0; i<256; i++)
   //   cout<<"hist["<<i<<"] ="<<hist[i]<<"   ";

   //ocurrence_cu(hist, image, width, height)   ;
   ocurrence_cu<<<image_num_blocks,image_num_threads>>>(g_image);
   //ocurrence_cu<<<image_num_blocks,image_num_threads>>>(g_image);
   hipDeviceSynchronize();
   //hipMemcpy(g_hist,       hist,        HistSize_Byte,         hipMemcpyHostToDevice);

   copy_data_from_shared<<<hist_num_blocks, hist_num_threads>>>(g_hist, gpu_Result, gpu_Block, gpu_Thread);



               hipMemcpy(cpu_Result, gpu_Result, HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Block,  gpu_Block,  HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Thread, gpu_Thread, HistSize_Byte, hipMemcpyDeviceToHost);
               //outputResult(cpu_Result, cpu_Block, cpu_Thread, 256);

   //for(int i=0; i<256; i++)
   //   cout<<"hist["<<i<<"] ="<<hist[i]<<"\n";

   nonZero_ocurrence_cu<<<hist_num_blocks, hist_num_threads>>>(gpu_Result, gpu_Block, gpu_Thread);

               hipMemcpy(cpu_Result, gpu_Result, HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Block,  gpu_Block,  HistSize_Byte, hipMemcpyDeviceToHost);
               hipMemcpy(cpu_Thread, gpu_Thread, HistSize_Byte, hipMemcpyDeviceToHost);
               outputResult(cpu_Result, cpu_Block, cpu_Thread, 256);












   minProp_cu<<<hist_num_blocks, hist_num_threads>>>(g_p, g_hist,g_width,g_height );
   //maxcodelen = MaxLength_cu(p) - 3;
   totalNode<<<1,1>>>(g_totalnodes,g_nodes);

   pix_freq  = (pixfreq<25>*)malloc(sizeof(pixfreq<25>) * totalnodes);
   huffcodes = (struct huffcode*)malloc(sizeof(struct huffcode) * nodes);

   hipMalloc((void **)&g_pix_freq,   sizeof(pixfreq<25>*) * totalnodes);
   hipMalloc((void **)&g_huffcodes,  sizeof(struct huffcode) * nodes);

   InitStruct_cu<<<hist_num_blocks, hist_num_threads>>>(g_pix_freq, g_huffcodes, g_hist, g_height, g_width);
/*
   hipMemcpy(image,        g_image,      IMAGE_SIZE_IN_BYTES,   hipMemcpyDeviceToHost);
   hipMemcpy(&width,       g_width,      sizeof(int),           hipMemcpyDeviceToHost);
   hipMemcpy(&height,      g_height,     sizeof(int),           hipMemcpyDeviceToHost);
   hipMemcpy(&hist,        g_hist,       HistSize*sizeof(int),  hipMemcpyDeviceToHost);
   hipMemcpy(&nodes,       g_nodes,      sizeof(int),           hipMemcpyDeviceToHost);
   hipMemcpy(&p,           g_p,          sizeof(int),           hipMemcpyDeviceToHost);
   hipMemcpy(&totalnodes,  g_totalnodes, sizeof(int),           hipMemcpyDeviceToHost);

*/
   sortHist_cu(huffcodes, nodes);
   BuildTree_cu(pix_freq, huffcodes, nodes);
   AssignCode_cu(pix_freq, nodes, totalnodes);
   PrintHuffmanCode(pix_freq, nodes);
   calBitLength(pix_freq, nodes);
   delete[] image; image = NULL;

   hipFree(g_image);
   hipFree(g_width);
   hipFree(g_height);
   hipFree(g_hist);
   hipFree(g_nodes);
   hipFree(g_totalnodes);
   hipFree(g_pix_freq);
   hipFree(g_huffcodes);
   hipFree(g_p);
   hipFree(g_MaxSize);
   hipFree(gpu_Result);
	hipFree(gpu_Block);
	hipFree(gpu_Thread);
   delete[] cpu_Result;
	delete[] cpu_Block;
	delete[] cpu_Thread;
   delete[] image2;
   delete[] hist;

}

int main()
{
   cout<<"Using Local CPU"<<endl;
   //compressionDriver();

   cout<<"\n\nUsing GPU"<<endl;
   compressionDriver_CL();

   return 0;

}

   // Encode the Image
   //int pix_val;
   //int l;

   // Writing the Huffman encoded
   // Image into a text file

   /*ofstream imagehuff; 
   imagehuff.open ("encoded_image.bin", ios::out | ios::app | ios::binary);
   cout<<"bien aqui<"<<endl;

   for (i = 0; i < height; i++)
   {
   for (j = 0; j < width; j++)
   {
      pix_val = image[i][j];
      cout<<"image[" <<i<<"]["<<j<<"] ="<<image[i][j]<<" ";
      for (l = 0; l < nodes; l++)
      {
         if (pix_val == pix_freq[l].intensity)
            imagehuff<< pix_freq[l].code;
      }
   }
   cout<<endl;
   }
   */