//
//  assignment3.cpp
//  assignment3
//
//  Created by Wilson on 2/10/22.
//



#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define ARRAY_SIZE 64
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

/* Declare  statically two arrays of ARRAY_SIZE each */
unsigned int cpu_arr1[ARRAY_SIZE];
unsigned int cpu_arr2[ARRAY_SIZE];
unsigned int cpu_result[ARRAY_SIZE];

__global__
void init(unsigned int *arr1, unsigned int *arr2, unsigned int *result)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	arr1[thread_idx] = thread_idx;
	arr2[thread_idx] = thread_idx % 3;
	result[thread_idx] = 0;
	
	
}
__global__
void add_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] + arr2[thread_idx];
	
	//block[thread_idx] = blockIdx.x;
	//thread[thread_idx] = threadIdx.x;
}

void main_sub0()
{

	/* Declare pointers for GPU based params */
	unsigned int *gpu_arr1;
	unsigned int *gpu_arr2;
	unsigned int *gpu_result;

	hipMalloc((void **)&gpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_result, ARRAY_SIZE_IN_BYTES);
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_result, gpu_result, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);

	const unsigned int numthread_per_block = 16;
	const unsigned int num_blocks = ARRAY_SIZE/numthread_per_block;
	const unsigned int num_threads = ARRAY_SIZE/num_blocks;

	/* Execute init kernel */
	init<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_result);
	
	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_result, gpu_result, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);


	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipFree(gpu_result);
	
	
	/* Iterate through the arrays and print */
	cout<<"######################################"<<endl;
	cout<<"blocks = "<<num_blocks<<"\tThreads = "<<num_threads<<endl;
	for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		cout<<"Array1["<<i<<"] = "<<cpu_arr1[i]
		<<"\tArray2["<<i<<"] = "<<cpu_arr1[i]
		<<"result["<<i<<"] = "<<cpu_result[i]<<endl;
	}
	cout<<"######################################"<<endl;

	/* Execute init kernel */
	//add_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_result);


	/* Iterate through the arrays and print */
	//for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	//{
	//	cout<<("Thread: %2u - Block: %2u\n",cpu_thread[i],cpu_block[i]);
	//}
}

int main()
{
	main_sub0();

	return EXIT_SUCCESS;
}
