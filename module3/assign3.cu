#include "hip/hip_runtime.h"
//
//  assignment3.cpp
//  assignment3
//
//  Created by Wilson on 2/10/22.
//


#include <iostream>
#include <time>

using namespace std;

#define ARRAY_SIZE 64
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
#define ARRAY_SIZE_IN_BYTES1 (sizeof(int) * (ARRAY_SIZE))


__global__
void init(unsigned int *arr1, unsigned int *arr2, 
		  unsigned int *r1, int *r2, unsigned int *r3, unsigned int *r4)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	arr1[thread_idx] = thread_idx;
	arr2[thread_idx] = thread_idx % 4;	
	
	r1[thread_idx]   = 0;
	r2[thread_idx]   = 0;
	r3[thread_idx]   = 0;
	r4[thread_idx]   = 0;

	
}
__global__
void add_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] + arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}


__global__
void sub_arr(unsigned int *arr1, unsigned int *arr2, int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] - arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

__global__
void mul_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] * arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

__global__
void mod_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(arr2[thread_idx]>0)
		result[thread_idx] = arr1[thread_idx] % arr2[thread_idx];
	else
		result[thread_idx] = 99999999;
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

void main_sub0()
{

	/* Declare  statically two arrays of ARRAY_SIZE each */
	unsigned int cpu_arr1[ARRAY_SIZE];
	unsigned int cpu_arr2[ARRAY_SIZE];
	unsigned int cpu_addResult[ARRAY_SIZE];
         	 int cpu_subResult[ARRAY_SIZE];
	unsigned int cpu_mulResult[ARRAY_SIZE];
	unsigned int cpu_modResult[ARRAY_SIZE];
	unsigned int cpu_addBlock[ARRAY_SIZE];
	unsigned int cpu_addThread[ARRAY_SIZE];	
	unsigned int cpu_subBlock[ARRAY_SIZE];
	unsigned int cpu_subThread[ARRAY_SIZE];	
	unsigned int cpu_mulBlock[ARRAY_SIZE];
	unsigned int cpu_mulThread[ARRAY_SIZE];	
	unsigned int cpu_modBlock[ARRAY_SIZE];
	unsigned int cpu_modThread[ARRAY_SIZE];	


	/* Declare pointers for GPU based params */
	unsigned int *gpu_arr1;
	unsigned int *gpu_arr2;
	unsigned int *gpu_addResult;
	         int *gpu_subResult;
	unsigned int *gpu_mulResult;
	unsigned int *gpu_modResult;
	unsigned *int gpu_addBlock[ARRAY_SIZE];
	unsigned *int gpu_addThread[ARRAY_SIZE];	
	unsigned *int gpu_subBlock[ARRAY_SIZE];
	unsigned *int gpu_subThread[ARRAY_SIZE];	
	unsigned *int gpu_mulBlock[ARRAY_SIZE];
	unsigned *int gpu_mulThread[ARRAY_SIZE];	
	unsigned *int gpu_modBlock[ARRAY_SIZE];
	unsigned *int gpu_modThread[ARRAY_SIZE];	

	hipMalloc((void **)&gpu_arr1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_subResult, ARRAY_SIZE_IN_BYTES1);
	hipMalloc((void **)&gpu_mulResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addBlock, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_subBlock, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_subThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_mulBlock, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_mulThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modBlock, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modThread, ARRAY_SIZE_IN_BYTES);
		
	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addResult, gpu_addResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subResult, gpu_subResult, ARRAY_SIZE_IN_BYTES1, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulResult, gpu_mulResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modResult, gpu_modResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addBlock, gpu_addBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addThread,gpu_addThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subBlock, gpu_subBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subThread,gpu_subThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulBlock, gpu_mulBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulThread,gpu_mulThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modBlock, gpu_modBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modThread,gpu_modThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);


	const unsigned int numthread_per_block = 16;
	const unsigned int num_blocks = ARRAY_SIZE/numthread_per_block;
	const unsigned int num_threads = ARRAY_SIZE/num_blocks;

	/* Execute init kernel */
	init<<<num_blocks, num_threads>>>(gpu_arr1,      gpu_arr2, 
									  gpu_addResult, gpu_subResult,
									  gpu_mulResult, gpu_modResult);
									  
	/* Execute init kernel */
	add_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_addResult, 
										 gpu_addBlock, gpu_addThread);
										 
	sub_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_subResult, 
										 gpu_subBlock, gpu_subThread);
										 	
	mul_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_mulResult, 
										 gpu_mulBlock, gpu_mulThread);
										 								                
	mod_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_modResult, 
										 gpu_modBlock, gpu_modThread);
										 
										  
	/* Free the arrays on the GPU as now we're done with them */

	hipMemcpy(cpu_arr1, gpu_arr1, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2, gpu_arr2, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addResult, gpu_addResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subResult, gpu_subResult, ARRAY_SIZE_IN_BYTES1, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulResult, gpu_mulResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modResult, gpu_modResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addBlock, gpu_addBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addThread, gpu_addThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subBlock, gpu_subBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subThread, gpu_subThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulBlock, gpu_mulBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulThread, gpu_mulThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modBlock, gpu_modBlock, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modThread, gpu_modThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);


	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipFree(gpu_addResult);
	hipFree(gpu_subResult);
	hipFree(gpu_mulResult);
	hipFree(gpu_modResult);
	hipFree(gpu_addBlock);
	hipFree(gpu_addThread);
	hipFree(gpu_subBlock);
	hipFree(gpu_subThread);
	hipFree(gpu_mulBlock);
	hipFree(gpu_mulThread);
	hipFree(gpu_modBlock);
	hipFree(gpu_modThread);
	
	
	
	/* Iterate through the arrays and print */
	cout<<"######################################"<<endl;
	cout<<"blocks = "<<num_blocks<<"\tThreads = "<<num_threads<<endl;
	for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		cout<<"Array1["<<i<<"] = "<<cpu_arr1[i]
		<<"\tArray2["<<i<<"] = "<<cpu_arr2[i]
		<<"\tresult["<<i<<"] = "<<cpu_modResult[i]<<endl;
	}
	cout<<"######################################"<<endl;

	


	/* Iterate through the arrays and print */
	//for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	//{
	//	cout<<("Thread: %2u - Block: %2u\n",cpu_thread[i],cpu_block[i]);
	//}
}

int main()
{
	main_sub0();

	return EXIT_SUCCESS;
}
