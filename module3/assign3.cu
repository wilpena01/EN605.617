//
//  assignment3.cpp
//  assignment3
//
//  Created by Wilson on 2/10/22.
//



#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define ARRAY_SIZE 256
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
#define ARRAY_SIZE_IN_BYTES1 (sizeof(int) * (ARRAY_SIZE))

__global__
void init(unsigned int *arr1, unsigned int *arr2, 
		  unsigned int *r1, int *r2, unsigned int *r3, unsigned int *r4)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	arr1[thread_idx] = thread_idx;
	arr2[thread_idx] = thread_idx % 4;	
	
	r1[thread_idx]   = 0;
	r2[thread_idx]   = 0;
	r3[thread_idx]   = 0;
	r4[thread_idx]   = 0;

	
}
__global__
void add_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] + arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}


__global__
void sub_arr(unsigned int *arr1, unsigned int *arr2, int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] - arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

__global__
void mul_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	result[thread_idx] = arr1[thread_idx] * arr2[thread_idx];
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

__global__
void mul_branch(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (thread_idx%2 == 0)
		result[thread_idx] = arr1[thread_idx] * arr2[thread_idx];
	else
		result[thread_idx] = 99999999;
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

__global__
void mod_arr(unsigned int *arr1, unsigned int *arr2, unsigned int *result,
			 unsigned int *block, unsigned int *thread)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(arr2[thread_idx] != 0)
		result[thread_idx] = arr1[thread_idx] % arr2[thread_idx];
	else
		result[thread_idx] = 99999999;
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;
}

int main()
{
	const unsigned int numthread_per_block = 64;
	const unsigned int num_blocks = ARRAY_SIZE/numthread_per_block;
	const unsigned int num_threads = ARRAY_SIZE/num_blocks;
	
	/* Declare  statically arrays of ARRAY_SIZE each */
	unsigned int cpu_arr1[ARRAY_SIZE];
	unsigned int cpu_arr2[ARRAY_SIZE];
	unsigned int cpu_addResult[ARRAY_SIZE];
	unsigned int cpu_addBlock[ARRAY_SIZE];
	unsigned int cpu_addThread[ARRAY_SIZE];	
         	 int cpu_subResult[ARRAY_SIZE];
    unsigned int cpu_subBlock[ARRAY_SIZE];
	unsigned int cpu_subThread[ARRAY_SIZE];	
	unsigned int cpu_mulResult[ARRAY_SIZE];
	unsigned int cpu_mulBlock[ARRAY_SIZE];
	unsigned int cpu_mulThread[ARRAY_SIZE];	
	unsigned int cpu_modResult[ARRAY_SIZE];
	unsigned int cpu_modBlock[ARRAY_SIZE];
	unsigned int cpu_modThread[ARRAY_SIZE];	
	unsigned int cpu_brResult[ARRAY_SIZE];
	unsigned int cpu_brBlock[ARRAY_SIZE];
	unsigned int cpu_brThread[ARRAY_SIZE];	
	
	/* Declare pointers for GPU based params */
	unsigned int *gpu_arr1;
	unsigned int *gpu_arr2;
	unsigned int *gpu_addResult;
	unsigned int *gpu_addBlock;
	unsigned int *gpu_addThread;
	         int *gpu_subResult;
	unsigned int *gpu_subBlock;
	unsigned int *gpu_subThread;
	unsigned int *gpu_mulResult;
	unsigned int *gpu_mulBlock;
	unsigned int *gpu_mulThread;	
	unsigned int *gpu_modResult;
	unsigned int *gpu_modBlock;
	unsigned int *gpu_modThread;	
	unsigned int *gpu_brResult;
	unsigned int *gpu_brBlock;
	unsigned int *gpu_brThread;		

	/* allocate memory for GPU based params */
	hipMalloc((void **)&gpu_arr1,      ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_arr2,      ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addBlock,  ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_addThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_subResult, ARRAY_SIZE_IN_BYTES1);
	hipMalloc((void **)&gpu_subBlock,  ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_subThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_mulResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_mulBlock,  ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_mulThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modResult, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modBlock,  ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_modThread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_brResult,  ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_brBlock,   ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_brThread,  ARRAY_SIZE_IN_BYTES);
	hipMemcpy(cpu_arr1,      gpu_arr1,      ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_arr2,      gpu_arr2,      ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addResult, gpu_addResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addBlock,  gpu_addBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_addThread, gpu_addThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subResult, gpu_subResult, ARRAY_SIZE_IN_BYTES1,hipMemcpyHostToDevice);
	hipMemcpy(cpu_subBlock,  gpu_subBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subThread, gpu_subThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulResult, gpu_mulResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulBlock,  gpu_mulBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_mulThread, gpu_mulThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modResult, gpu_modResult, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modBlock,  gpu_modBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modThread, gpu_modThread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_subResult, gpu_brResult,  ARRAY_SIZE_IN_BYTES1,hipMemcpyHostToDevice);
	hipMemcpy(cpu_modBlock,  gpu_brBlock,   ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(cpu_modThread, gpu_brThread,  ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);



 	
 	
	/* Execute kernels */
	init<<<num_blocks, num_threads>>>(gpu_arr1,      gpu_arr2, 
									  gpu_addResult, gpu_subResult,
									  gpu_mulResult, gpu_modResult);
									  
	add_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_addResult, 
										 gpu_addBlock, gpu_addThread);
										 
	sub_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_subResult, 
										 gpu_subBlock, gpu_subThread);
										 
	auto start1 = high_resolution_clock::now();									 	
	mul_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_mulResult, 
										 gpu_mulBlock, gpu_mulThread);
	auto stop1 = high_resolution_clock::now();	
	auto start2 = high_resolution_clock::now();									 
	mul_branch<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_brResult, 
										 gpu_brBlock, gpu_brThread);
	
	auto stop2 = high_resolution_clock::now();									 								                
	mod_arr<<<num_blocks, num_threads>>>(gpu_arr1, gpu_arr2, gpu_modResult, 
										 gpu_modBlock, gpu_modThread);
	hipDeviceSynchronize();
										 
	
	
	auto duration1 = duration_cast<microseconds>(stop1 - start1);
	auto duration2 = duration_cast<microseconds>(stop2 - start2);
	  
	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy(cpu_arr1,      gpu_arr1,      ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_arr2,      gpu_arr2,      ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addResult, gpu_addResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addBlock,  gpu_addBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_addThread, gpu_addThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subResult, gpu_subResult, ARRAY_SIZE_IN_BYTES1,hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subBlock,  gpu_subBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_subThread, gpu_subThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulResult, gpu_mulResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulBlock,  gpu_mulBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_mulThread, gpu_mulThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modResult, gpu_modResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modBlock,  gpu_modBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_modThread, gpu_modThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_brResult, gpu_brResult, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_brBlock,  gpu_brBlock,  ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_brThread, gpu_brThread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipFree(gpu_arr1);
	hipFree(gpu_arr2);
	hipFree(gpu_addResult);
	hipFree(gpu_addBlock);
	hipFree(gpu_addThread);
	hipFree(gpu_subResult);
	hipFree(gpu_subBlock);
	hipFree(gpu_subThread);
	hipFree(gpu_mulResult);
	hipFree(gpu_mulBlock);
	hipFree(gpu_mulThread);
	hipFree(gpu_modResult);
	hipFree(gpu_modBlock);
	hipFree(gpu_modThread);
	hipFree(gpu_brResult);
	hipFree(gpu_brBlock);
	hipFree(gpu_brThread);
	
	
	ofstream output("out1.txt", std::ofstream::out);
	
	/* Iterate through the arrays and print 
	cout<<"\nTotal # of Threads = "<<ARRAY_SIZE
	      <<"\nNumber of threads per block = "<<numthread_per_block
	      <<"\nTotal # of blocks = "<<num_blocks
	      <<"\nElapsed Mul time is = "<< ms.count() << " milliseconds\n"
	      <<"\nElapsed time is = "<< ms.count() << " milliseconds\n"
	      <<"\n######################################\n";*/
	
	for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		cout<<"Array1["<<i<<"] = "<<cpu_arr1[i]<<"\nArray2["<<i<<"] = "<<cpu_arr2[i]
		
		<<"\nAdd["<<i<<"] = "<<cpu_addResult[i]<<"\taddBock["<<i<<"] = "<<cpu_addBlock[i]
		<<"\taddThread["<<i<<"] = "<<cpu_addThread[i]<<"\n"
		
		
		<<"Sub["<<i<<"] = "<<cpu_subResult[i]<<"\tsubBock["<<i<<"] = "<<cpu_subBlock[i]
		<<"\tsubThread["<<i<<"] = "<<cpu_subThread[i]<<"\n"
		
		
		<<"Mul["<<i<<"] = "<<cpu_mulResult[i]<<"\tmulBock["<<i<<"] = "<<cpu_mulBlock[i]
		<<"\tmulThread["<<i<<"] = "<<cpu_mulThread[i]<<"\n"
		
		
		<<"Mod["<<i<<"] = "<<cpu_modResult[i]<<"\tmodBock["<<i<<"] = "<<cpu_modBlock[i]
		<<"\tmodThread["<<i<<"] = "<<cpu_modThread[i]<<"\n"
		
		
		<<"MulB["<<i<<"] = "<<cpu_brResult[i]<<"\tBr_Bock["<<i<<"] = "<<cpu_brBlock[i]
		<<"\tBr_Thread["<<i<<"] = "<<cpu_brThread[i]<<"\n"
		
		<<"\n######################################\n";

	}
	
	output.close();

		/* Iterate through the arrays and print */
	cout<<"\nTotal # of Threads = "<<ARRAY_SIZE
	      <<"\nNumber of threads per block = "<<numthread_per_block
	      <<"\nTotal # of blocks = "<<num_blocks
	      <<"\nElapsed Mul time is = "<< duration1.count() << " milliseconds"
	      <<"\nElapsed Mul Branched time is = "<< duration2.count() << " milliseconds\n"
	      <<"\n######################################\n";


	/* Iterate through the arrays and print */
	//for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	//{
	//	cout<<("Thread: %2u - Block: %2u\n",cpu_thread[i],cpu_block[i]);
	//}
	
	
	return EXIT_SUCCESS;
}
